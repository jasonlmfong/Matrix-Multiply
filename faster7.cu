#include "hip/hip_runtime.h"
#include <stdio.h>

#define N  4096

__global__ void matrixMulGPU( int * a, int * b, int * c )
{
  /*
   * Build out this kernel.
   */
  
  int val = 0;
  
  int rowIndexWithinTheGrid = threadIdx.x + blockIdx.x * blockDim.x;
  int rowGridStride = gridDim.x * blockDim.x;
  
  int colIndexWithinTheGrid = threadIdx.y + blockIdx.y * blockDim.y;
  int colGridStride = gridDim.y * blockDim.y;
  
  for(int row = rowIndexWithinTheGrid; row < N; row += rowGridStride)
  {
    for(int col = colIndexWithinTheGrid; col < N; col += colGridStride)
    {
      val = 0;
      for ( int k = 0; k < N; ++k )
        val += a[row * N + k] * b[k * N + col];
      c[row * N + col] = val;
    }
  }
}

/*
 * This CPU function already works, and will run to create a solution matrix
 * against which to verify your work building out the matrixMulGPU kernel.
 */

int main()
{
  int deviceId;
  int numberOfSMs;

  hipGetDevice(&deviceId);
  hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);

  int *a, *b, *c_gpu; // Allocate a solution matrix for both the CPU and the GPU operations

  int size = N * N * sizeof (int); // Number of bytes of an N x N matrix

  // Allocate memory
  hipMallocManaged (&a, size);
  hipMallocManaged (&b, size);
  hipMallocManaged (&c_gpu, size);

  // Initialize memory; create sample 2D matrices
  for( int row = 0; row < N; ++row )
    for( int col = 0; col < N; ++col )
    {
      a[row*N + col] = row;
      b[row*N + col] = col+2;
      c_gpu[row*N + col] = 0;
    }

  // prefecth to GPU memory for operation
  hipMemPrefetchAsync(a, size, deviceId);
  hipMemPrefetchAsync(b, size, deviceId);
  hipMemPrefetchAsync(c_gpu, size, deviceId);
  
  /*
   * Assign `threads_per_block` and `number_of_blocks` 2D values
   * that can be used in matrixMulGPU above.
   */

  dim3 number_of_blocks((numberOfSMs * 32), (numberOfSMs * 32), 1);
  dim3 threads_per_block(32, 32, 1);
  
  matrixMulGPU <<< number_of_blocks, threads_per_block >>> ( a, b, c_gpu );

  hipDeviceSynchronize(); // wait for it tofinish

  printf("Success!\n");

  // Free all our allocated memory
  hipFree(a); hipFree(b);
  hipFree( c_gpu );
}
